#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <math.h>
#include <algorithm>
#include <stdint.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
#include "hd_structs.cuh"
#include "cuda_common.cuh"
#include "cpp_functions.h"
#include "cuda_functions.cuh"



int main() {

	int sizeTrack = 0;
	int block_size = 64;


	int simGPU = 1;

    //Alocando memoria para atributos das structs
	inicializarStructs();
    
	//Lendo os arquivos de entrada e inicializando os pacotes de simulação.
	pmrdr2_();


	if (simGPU){//Simulação na GPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;
		//Resete da GPU
		gpuErrchk(hipDeviceReset());
		//gpuErrchk(hipSetDevice(0));
		//gpuErrchk(hipDeviceSynchronize());
		//gpuErrchk(hipDeviceSynchronize());

		//aloca vetores das particulas primarias e secundarias

	
		PRITRACK =  (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas primarias
		SECTRACK_G = (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas secundarias de fotons
		SECTRACK_E = (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas secudarias de eletrons
		SECTRACK_P = (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas secundarias de protons
		gpuErrchk(hipMalloc((void **)&d_TRACK_mod, sizeof(hd_TRACK_MOD)*pilhaPart));

		bool btransfCPU_to_GPU = false;
		
		

		while ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)){

			
			//criar vetor de particulas primarias inicial
			iniPRITRACK(); 

			//transferindo os structs da CPU para GPU
			if (!btransfCPU_to_GPU){
				transfCPU_to_GPU();
				btransfCPU_to_GPU = true;
			}
			//seta o tamanho da pilha a ser simulada
			sizeTrack = pilhaPart;

			//transfere a pilha de particulas secundarias para GPU
			transfSecTracksCPU_to_GPU();

			printf("aqui\n");

			//transfere as particulas a serem simuladas para a GPU
    		gpuErrchk(hipMemcpy(d_TRACK_mod, PRITRACK, sizeof(hd_TRACK_MOD)*pilhaPart, hipMemcpyHostToDevice));
    		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), d_TRACK_mod, sizeof(hd_TRACK_MOD)*pilhaPart));
			
			//Quantidade de blocos no grid e de threads nos blocos
			dim3 block(block_size);
			dim3 grid((sizeTrack / block.x));

			printf("%lf\n", PRITRACK[0].E);
			printf("%lf\n", PRITRACK[1].E);

			//chamada do kernel para simulacao das particulas primarias
			showers<<<grid,block>>>(sizeTrack);

			//Aguarda o termino da simulação das particulas primarias enviadas
			gpuErrchk(hipDeviceSynchronize());

			//resgata o pacote de particulas primarias da gpu
			transfSecTracksGPU_to_CPU();

		
			printf("\nquantidade de parricula secundaria photon %d\n\n", nTRACKS_.nSECTRACK_G);
			printf("\nquantidade de parricula secundaria eletron %d\n\n", nTRACKS_.nSECTRACK_E);
			printf("\nquantidade de parricula secundaria positron %d\n\n", nTRACKS_.nSECTRACK_P);
			//while para realizar simulação das particulas secundarias
			//verifica se o pacote de particulas secundarias esta grande o suficinete para uma simulacao
			//Se for grande o suficiente, ondena o vetor de particulas secundarias
			//chama a simulacao das particulas secundarias se for grande o suficiente	
			//	showers<<<1,1>>>();

			if (*CSOUR0_.JOBEND != 0)
				goto L202;

			timer2_(*CNTRL_.TSEC);

			//verifica tempo do DUMP
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					//retorna os dados da GPU para imprimir o DUMP
					transfGPU_to_CPU();
					memoryFreeGPU();
					hipDeviceSynchronize();
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
				}
			}
		}

L202:;
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;

		//retorna os dados da GPU para imprimir o DUMP
		transfGPU_to_CPU();
		memoryFreeGPU();
		gpuErrchk(hipFree(d_TRACK_mod));
					free(PRITRACK);
			free(SECTRACK_G);
			free(SECTRACK_E);
			free(SECTRACK_P);
		


	}else{ //Simulação na CPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;

L101:;
		//Simulação de uma nova ducha e pontuação.
		shower2_();
		if (*CSOUR0_.JOBEND != 0)
			goto L102;

		timer2_(*CNTRL_.TSEC);

		//Terminar a simulação após o tempo previsto ou após completar Chuveiros DSHN.
		if ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)) {
			//Escreva os resultados parciais após cada período de despejo.
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
					goto L101;
				}
			}
			goto L101;
		}
L102:;
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
	}

L103:;//Imprimir resultados Finais
		pmwrt2_(1);
		printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
		plotdose2_();
		memoryFree();
		printf("  *** END ***\n");
		return 0;
	
}

