#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <math.h>
#include <algorithm>
#include <stdint.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
#include "hd_structs.cuh"
#include "cuda_common.cuh"
#include "cpp_functions.h"
#include "cuda_functions.cuh"



int main() {

	int simGPU = 1;

    //Alocando memoria para atributos das structs
	inicializarStructs();
    
	//Lendo os arquivos de entrada e inicializando os pacotes de simulação.
	pmrdr2_();


	if (simGPU){//Simulação na GPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;
		//Resete da GPU
		hipDeviceReset();

		//aloca vetores das particulas primarias e secundarias

	
		PRITRACK =  (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas primarias
		SECTRACK_G = (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas secundarias de fotons
		SECTRACK_E = (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas secudarias de eletrons
		SECTRACK_P = (hd_TRACK_MOD*)malloc(pilhaPart * sizeof(hd_TRACK_MOD)); //vetor de particulas secundarias de protons


		//transferindo os structs da CPU para GPU
		transfCPU_to_GPU();

		while ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)){

			//criar vetor de particulas primarias
			iniPRITRACK(); 
			//transfere a pilha de particulas secundarias para GPU
			transfSecTracksCPU_to_GPU();
			//chamada do kernel para simulacao das particulas primarias
		//	showers<<<1,1>>>();
			//Aguarda o termino da simulação das particulas primarias enviadas
			hipDeviceSynchronize();
			//resgata o pacote de particulas primarias da gpu
			transfSecTracksGPU_to_CPU();
			//while para realizar simulação das particulas secundarias
			//verifica se o pacote de particulas secundarias esta grande o suficinete para uma simulacao
			//Se for grande o suficiente, ondena o vetor de particulas secundarias
			//chama a simulacao das particulas secundarias se for grande o suficiente	
			//	showers<<<1,1>>>();

			//verifica tempo do DUMP
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					//retorna os dados da GPU para imprimir o DUMP
					transfGPU_to_CPU();
					hipDeviceSynchronize();
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
				}
			}
		}
	
		memoryFreeGPU();
		free(PRITRACK);
		free(SECTRACK_G);
		free(SECTRACK_E);
		free(SECTRACK_P);


	}else{ //Simulação na CPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;

L101:;
		//Simulação de uma nova ducha e pontuação.
		shower2_();
		if (*CSOUR0_.JOBEND != 0)
			goto L102;

		timer2_(*CNTRL_.TSEC);

		//Terminar a simulação após o tempo previsto ou após completar Chuveiros DSHN.
		if ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)) {
			//Escreva os resultados parciais após cada período de despejo.
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
					goto L101;
				}
			}
			goto L101;
		}
L102:;
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
	}

L103:;//Imprimir resultados Finais
		pmwrt2_(1);
		printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
		plotdose2_();
		transfCPU_to_GPU();
		hipDeviceSynchronize();
		transfGPU_to_CPU();
		memoryFreeGPU();
		hipDeviceSynchronize();
		memoryFree();
		printf("  *** END ***\n");
		return 0;
	

}

