#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <math.h>
#include <algorithm>
#include <stdint.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
using namespace std;
#include "hd_structs.cuh"
#include "ISEEDS.h"
#include "cuda_common.cuh"
#include "cuda_functions.cuh"
#include "cpp_functions.h"


int main() {

	//Alocando memoria para atributos das structs
	memoryAllocCPU();

	
	int simGPU = 1;

	//Lendo os arquivos de entrada e inicializando os pacotes de simulação.
	pmrdr2_();

	if (simGPU){//Simulação na GPU

		int sizeTrack = 0;

		if (*CSOUR0_.JOBEND != 0)
			goto L103;

		//alocando memooria na GPU
		memoryAllocGPU();

		//Inicializa as semnentes para GPU e par CPU
		initializeISSEDS_();
		*RSEED_.ISEED1 = IS1[0];
		*RSEED_.ISEED2 = IS2[0];

		sizeTrack = pilhaPart;

		dim3 block(blockSize);
		dim3 grid(ceil(sizeTrack / block.x)+1);

		//inicializa gerando de numeros aleatorios cuRand
		initializeRand<<<grid, block>>>(sizeTrack);
		gpuErrchk(hipDeviceSynchronize());

		//trasnferindo dados para GPU
		transfCPU_to_GPU();

		while ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)){
			
			cleans2GPU_();
			simPriTrack_G();
			transfnTRACKSGPU_to_CPU();

			// simulação de particulas secundarias
			while ((nTRACKS_.nSECTRACK_E > 0) || (nTRACKS_.nSECTRACK_G > 0) || (nTRACKS_.nSECTRACK_P > 0))
			{
				if (nTRACKS_.nSECTRACK_E > 0)
				{
					simSecTrack_E();
				}

				if (nTRACKS_.nSECTRACK_P > 0)
				{
					simSecTrack_P();
				}

				if (nTRACKS_.nSECTRACK_G > 0)
				{
					simSecTrack_G();
				}
			}
			gpuErrchk(hipDeviceSynchronize());

			//contabilizacao da contribuição das particulas para a dose geral
			showers_cont<<<grid,block>>>(sizeTrack);
			gpuErrchk(hipDeviceSynchronize());

			printf("Simulado: %f\n", *CNTRL_.SHN);
			
			if (*CSOUR0_.JOBEND != 0)
				goto L202;

			timer2_(*CNTRL_.TSEC);

			//verifica tempo do DUMP
			//if (*CDUMP_.LDUMP) {
				if ((*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) || (*CNTRL_.SHN == 1e4) || (*CNTRL_.SHN == 1e5) || (*CNTRL_.SHN == 1e6) || (*CNTRL_.SHN == 1e7) || (*CNTRL_.SHN == 1e8) || (*CNTRL_.SHN == 1e9)) {
				
					//retorna os dados da gpu para cpu
					transfGPU_to_CPU();
					gpuErrchk(hipDeviceSynchronize());

					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;

					//imprime os resultados parciais da simulação
					pmwrt2_(1);
					plotdose2_();

					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();

					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
				}
			//}
		}


L202:;
	
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
		
		//retorna os dados da GPU para imprimir o DUMP
		transfGPU_to_CPU();
		
	}else{ //Simulação na CPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;

L101:;
		//Simulação de uma nova ducha e pontuação.
		shower2_();
		if (*CSOUR0_.JOBEND != 0)
			goto L102;

		timer2_(*CNTRL_.TSEC);

		//Terminar a simulação após o tempo previsto ou após completar Chuveiros DSHN.
		if ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)) {
			//Escreva os resultados parciais após cada período de despejo.
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
					goto L101;
				}
			}
			goto L101;
		}
L102:;
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
	}

L103:;//Imprimir resultados Finais
	printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
	if (simGPU == 0){
		pmwrt2_(1);
		plotdose2_();
	}else{
		memoryFreeGPU();
		gpuErrchk(hipDeviceReset());
	}
	memoryFreeCPU();
	printf("  *** END ***\n");
	return 0;
}

