#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <math.h>
#include <algorithm>
#include <stdint.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
#include "hd_structs.cuh"
#include "cpp_functions.h"
#include "cuda_common.cuh"
#include "cuda_functions.cuh"

int main() {

    //Alocando memoria para atributos das structs
	inicializarStructs();
    
	//Lendo os arquivos de entrada e inicializando os pacotes de simulação.
	pmrdr2_();

	if (*CSOUR0_.JOBEND != 0)
		goto L103;

L101:;
	//Simulação de uma nova ducha e pontuação.
	shower2_();
	if (*CSOUR0_.JOBEND != 0)
		goto L102;

	timer2_(*CNTRL_.TSEC);

	//Terminar a simulação após o tempo previsto ou após completar Chuveiros DSHN.
	if ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)) {
		//Escreva os resultados parciais após cada período de despejo.
		if (*CDUMP_.LDUMP) {
			if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
				*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
				pmwrt2_(-1);
				printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
				*CNTRL_.TSECAD = *CNTRL_.TSEC;
				*CNTRL_.CPUT0 = cputim2_();
				goto L101;
			}
		}
		goto L101;
	}

L102:;
	*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
L103:;
	pmwrt2_(1);
	printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
	plotdose2_();
//Resete da GPU
    hipDeviceReset();

    //transferindo os structs da CPU para GPU
    transfCPU_to_GPU();

    //teste<<<1,1>>>(); CHAMADA DA FUNCAO KERNEL

    hipDeviceSynchronize();

    transfGPU_to_CPU();

    memoryFreeGPU();
    memoryFree();
	printf("  *** END ***\n");
	return 0;
}

