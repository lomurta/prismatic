#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <math.h>
#include <algorithm>
#include <stdint.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
#include "hd_structs.cuh"
#include "cuda_common.cuh"
#include "cuda_functions.cuh"
#include "cpp_functions.h"




int main() {

	int sizeTrack = 0;
	int simGPU = 1;

    //Alocando memoria para atributos das structs
	memoryAllocCPU();
    
	//Lendo os arquivos de entrada e inicializando os pacotes de simulação.
	pmrdr2_();

	if (simGPU){//Simulação na GPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;
		//Resete da GPUFF
		//gpuErrchk(hipDeviceReset());

		//alocando memooria na GPU
		memoryAllocGPU();

		//aloca vetores das particulas primarias e secundarias
		bool btransfCPU_to_GPU = false;
		//bool btransfGPU_to_CPU = false;
		cleans2GPU_();
		
		while ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)){
			
			//criar vetor de particulas primarias inicial
			iniPRITRACK(); 
			quickSort(PRITRACK, 0, pilhaPart - 1);

			//transferindo os structs da CPU para GPU
			if (!btransfCPU_to_GPU){
				transfCPU_to_GPU();
				btransfCPU_to_GPU = true;
			}
			//seta o tamanho da pilha a ser simulada
			sizeTrack = pilhaPart;

			//transfere a pilha de particulas secundarias para GPU
			transfSecTracksCPU_to_GPU();

			//transfere as particulas a serem simuladas para a GPU
			//gpuErrchk(hipMalloc(&d_TRACK_mod, sizeof(hd_TRACK_MOD)*pilhaPart));
    		//gpuErrchk(hipMemcpy(d_TRACK_mod, PRITRACK, sizeof(hd_TRACK_MOD)*pilhaPart, hipMemcpyHostToDevice));
    		//gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), d_TRACK_mod, sizeof(hd_TRACK_MOD*)*pilhaPart));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), PRITRACK, sizeof(hd_TRACK_MOD)*pilhaPart,0));
			
			//Quantidade de blocos no grid e de threads nos blocos
			dim3 block(blockSize);
			dim3 grid(ceil(sizeTrack / block.x)+1);

			/*printf("[0]: %lf\n", PRITRACK[0].E);
			printf("[1]: %lf\n", PRITRACK[1].E);
			printf("[256]: %lf\n", PRITRACK[256].E);
			printf("[4095]: %lf\n", PRITRACK[4095].E);*/
			//printf("[256]: %lf\n", PRITRACK[4096].E);
			//chamada do kernel para simulacao das particulas primarias
			showers_pri<<<grid,block>>>(sizeTrack);
			//Aguarda o termino da simulação das particulas primarias enviadas
			gpuErrchk(hipDeviceSynchronize());
			//gpuErrchk(hipFree(d_TRACK_mod));

			//resgata o pacote de particulas primarias da gpu
			transfSecTracksGPU_to_CPU();
			
		
			//printf("\nquantidade de parricula secundaria photon %d\n\n", nTRACKS_.nSECTRACK_G);
			//printf("\nquantidade de parricula secundaria eletron %d\n\n", nTRACKS_.nSECTRACK_E);
			//printf("\nquantidade de parricula secundaria positron %d\n\n", nTRACKS_.nSECTRACK_P);
			//Esvaziando parcialmente particulas secundarias.
			/*if (nTRACKS_.nSECTRACK_E >= pilhaPart/2){
				while (nTRACKS_.nSECTRACK_E > pilhaPart/4){
					simSecTrack_E();
				}
			}
			if (nTRACKS_.nSECTRACK_G >= pilhaPart/2){
				while (nTRACKS_.nSECTRACK_G > pilhaPart/4){
					simSecTrack_G();
				}
			}
			if (nTRACKS_.nSECTRACK_P >= pilhaPart/2){
				while (nTRACKS_.nSECTRACK_P > pilhaPart/4){
					simSecTrack_P();
				}
			}*/

			//Zerando particulas segundarias
			while ((nTRACKS_.nSECTRACK_E > 0) || (nTRACKS_.nSECTRACK_G > 0) || (nTRACKS_.nSECTRACK_P > 0)){
			if (nTRACKS_.nSECTRACK_E > 0)
				simSecTrack_E();
			if (nTRACKS_.nSECTRACK_G > 0)
				simSecTrack_G();
			if (nTRACKS_.nSECTRACK_P > 0)
				simSecTrack_P();
			}
			gpuErrchk(hipDeviceSynchronize());

			sizeTrack = pilhaPart;
			//Quantidade de blocos no grid e de threads nos blocos
			dim3 blockCont(blockSize);
			dim3 gridCont(ceil(sizeTrack / block.x)+1);
			
			showers_cont<<<gridCont,blockCont>>>(sizeTrack);
			//Aguarda o termino da simulação das particulas primarias enviadas
			gpuErrchk(hipDeviceSynchronize());

			printf("Simulado: %f\n", *CNTRL_.SHN);


			
		/*	if ((nTRACKS_.nSECTRACK_E >= pilhaSec/2) || (nTRACKS_.nSECTRACK_G >= pilhaSec/2) || (nTRACKS_.nSECTRACK_P >= pilhaSec/2)){
				while ((nTRACKS_.nSECTRACK_E > 0) || (nTRACKS_.nSECTRACK_G > 0) || (nTRACKS_.nSECTRACK_P > 0 )){
				/*	printf("Antes da simulacao\n");
					printf("Quantidade de parricula secundaria photon: %d\n", nTRACKS_.nSECTRACK_G);
					printf("Quantidade de parricula secundaria eletron: %d\n", nTRACKS_.nSECTRACK_E);
					printf("Quantidade de parricula secundaria positron: %d\n\n", nTRACKS_.nSECTRACK_P);*/
				////	if (nTRACKS_.nSECTRACK_E > 0){
				//		simSecTrack_E();
				/*	printf("Apos simular Eletrons\n");
					printf("Quantidade de parricula secundaria photon: %d\n", nTRACKS_.nSECTRACK_G);
					printf("Quantidade de parricula secundaria eletron: %d\n", nTRACKS_.nSECTRACK_E);
					printf("Quantidade de parricula secundaria positron: %d\n\n", nTRACKS_.nSECTRACK_P);*/
				//	}
				//	if (nTRACKS_.nSECTRACK_G > 0){
				//		simSecTrack_G();
				/*	printf("Apos simular Fotons\n");
					printf("Quantidade de parricula secundaria photon: %d\n", nTRACKS_.nSECTRACK_G);
					printf("Quantidade de parricula secundaria eletron: %d\n", nTRACKS_.nSECTRACK_E);
					printf("Quantidade de parricula secundaria positron: %d\n\n", nTRACKS_.nSECTRACK_P);*/
					//}
				//	if (nTRACKS_.nSECTRACK_P > 0){
				//		simSecTrack_P();
				/*	printf("Apos simular Positrons\n");
					printf("Quantidade de parricula secundaria photon: %d\n", nTRACKS_.nSECTRACK_G);
					printf("Quantidade de parricula secundaria eletron: %d\n", nTRACKS_.nSECTRACK_E);
					printf("Quantidade de parricula secundaria positron: %d\n\n", nTRACKS_.nSECTRACK_P);*/
				//	}
			//	}
		//	}*/

			

			if (*CSOUR0_.JOBEND != 0)
				goto L202;

			timer2_(*CNTRL_.TSEC);

			//verifica tempo do DUMP
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					//retorna os dados da GPU para imprimir o DUMP
					
					transfGPU_to_CPU();
					gpuErrchk(hipDeviceSynchronize());
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
				}
			}
		}

		//Zerando particulas segundarias
		/*while ((nTRACKS_.nSECTRACK_E > 0) || (nTRACKS_.nSECTRACK_G > 0) || (nTRACKS_.nSECTRACK_P > 0 )){
			if (nTRACKS_.nSECTRACK_E > 0)
				simSecTrack_E();
			if (nTRACKS_.nSECTRACK_G > 0)
				simSecTrack_G();
			if (nTRACKS_.nSECTRACK_P > 0)
				simSecTrack_P();
		}*/

L202:;
		
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
		//retorna os dados da GPU para imprimir o DUMP
		transfGPU_to_CPU();
		gpuErrchk(hipDeviceSynchronize());
		memoryFreeGPU();
	}else{ //Simulação na CPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;

L101:;
		//Simulação de uma nova ducha e pontuação.
		shower2_();
		if (*CSOUR0_.JOBEND != 0)
			goto L102;

		timer2_(*CNTRL_.TSEC);

		//Terminar a simulação após o tempo previsto ou após completar Chuveiros DSHN.
		if ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)) {
			//Escreva os resultados parciais após cada período de despejo.
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
					goto L101;
				}
			}
			goto L101;
		}
L102:;
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
	}

L103:;//Imprimir resultados Finais
	printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
	pmwrt2_(1);
	
	plotdose2_();
	memoryFreeCPU();
	printf("  *** END ***\n");
	return 0;
	
}

