#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <math.h>
#include <algorithm>
#include <stdint.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
#include "hd_structs.cuh"
#include "cuda_common.cuh"
#include "cpp_functions.h"
#include "cuda_functions.cuh"



int main() {

	int sizeTrack = 0;
	int block_size = 128;
	int simGPU = 1;

    //Alocando memoria para atributos das structs
	memoryAllocCPU();
    
	//Lendo os arquivos de entrada e inicializando os pacotes de simulação.
	pmrdr2_();

	if (simGPU){//Simulação na GPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;
		//Resete da GPUFF
		//gpuErrchk(hipDeviceReset());

		//alocando memooria na GPU
		memoryAllocGPU();

		//aloca vetores das particulas primarias e secundarias
		bool btransfCPU_to_GPU = false;
		//bool btransfGPU_to_CPU = false;
		cleans2GPU_();
		
		while ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)){
			
			//criar vetor de particulas primarias inicial
			iniPRITRACK(); 

			//transferindo os structs da CPU para GPU
			if (!btransfCPU_to_GPU){
				transfCPU_to_GPU();
				btransfCPU_to_GPU = true;
			}
			//seta o tamanho da pilha a ser simulada
			sizeTrack = pilhaPart;

			//transfere a pilha de particulas secundarias para GPU
			transfSecTracksCPU_to_GPU();

			//transfere as particulas a serem simuladas para a GPU
			gpuErrchk(hipMalloc(&d_TRACK_mod, sizeof(hd_TRACK_MOD)*pilhaPart));
    		//gpuErrchk(hipMemcpy(d_TRACK_mod, PRITRACK, sizeof(hd_TRACK_MOD)*pilhaPart, hipMemcpyHostToDevice));
    		//gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), d_TRACK_mod, sizeof(hd_TRACK_MOD*)*pilhaPart));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), PRITRACK, sizeof(hd_TRACK_MOD)*pilhaPart,0));
			
			//Quantidade de blocos no grid e de threads nos blocos
			dim3 block(block_size);
			dim3 grid((sizeTrack / block.x));

			printf("[0]: %lf\n", PRITRACK[0].E);
			printf("[1]: %lf\n", PRITRACK[1].E);
			printf("[256]: %lf\n", PRITRACK[256].E);
			printf("[4095]: %lf\n", PRITRACK[4095].E);
			//printf("[256]: %lf\n", PRITRACK[4096].E);
			//chamada do kernel para simulacao das particulas primarias
			showers_pri<<<grid,block>>>(sizeTrack);
			//Aguarda o termino da simulação das particulas primarias enviadas
			gpuErrchk(hipDeviceSynchronize());
			gpuErrchk(hipFree(d_TRACK_mod));

			//resgata o pacote de particulas primarias da gpu
			transfSecTracksGPU_to_CPU();
		
			printf("\nquantidade de parricula secundaria photon %d\n\n", nTRACKS_.nSECTRACK_G);
			printf("\nquantidade de parricula secundaria eletron %d\n\n", nTRACKS_.nSECTRACK_E);
			printf("\nquantidade de parricula secundaria positron %d\n\n", nTRACKS_.nSECTRACK_P);
			
			if (nTRACKS_.nSECTRACK_E == pilhaSec){
				while (nTRACKS_.nSECTRACK_E > 1024){
					nTRACKS_.nSECTRACK_E = 0;
					sizeTrack = pilhaSec;

					//transfere a pilha de particulas secundarias para GPU
					transfSecTracksCPU_to_GPU();
					
					
					//transfere as particulas a serem simuladas para a GPU
					gpuErrchk(hipMalloc(&d_TRACK_mod, sizeof(hd_TRACK_MOD)*pilhaSec));
					//PRITRACK = SECTRACK_E;
					//gpuErrchk(hipMemcpy(d_TRACK_mod, SECTRACK_E, sizeof(hd_TRACK_MOD)*pilhaSec, hipMemcpyHostToDevice));
					//gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), d_TRACK_mod, sizeof(hd_TRACK_MOD)*pilhaSec));

					//memcpy(vTrack_Simular, SECTRACK_E, sizeof(hd_TRACK_MOD)*pilhaPart);

					gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dg_TRACK_mod_), SECTRACK_E, sizeof(hd_TRACK_MOD)*pilhaPart,0));

					
				

					dim3 blockSec(block_size);
					dim3 gridSec((sizeTrack / block.x));

					printf("[0]: %lf\n", SECTRACK_E[0].E);
					printf("[1]: %lf\n", SECTRACK_E[1].E);
					printf("[256]: %lf\n", SECTRACK_E[256].E);
					printf("[4095]: %lf\n", SECTRACK_E[4095].E);
					printf("[4096]: %lf\n", SECTRACK_E[4096].E);

					//chamada do kernel para simulacao das particulas primarias
					showers_sec<<<gridSec,blockSec>>>(sizeTrack);
						//Aguarda o termino da simulação das particulas primarias enviadas
					gpuErrchk(hipDeviceSynchronize());
						//resgata o pacote de particulas primarias da gpu
					gpuErrchk(hipFree(d_TRACK_mod));

					transfSecTracksGPU_to_CPU();
				}
			}

			if (*CSOUR0_.JOBEND != 0)
				goto L202;

			timer2_(*CNTRL_.TSEC);

			//verifica tempo do DUMP
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					//retorna os dados da GPU para imprimir o DUMP
					
					transfGPU_to_CPU();
					gpuErrchk(hipDeviceSynchronize());
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
				}
			}
		}

L202:;
		
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
		//retorna os dados da GPU para imprimir o DUMP
		transfGPU_to_CPU();
		gpuErrchk(hipDeviceSynchronize());
		memoryFreeGPU();
		printf("aqui\n");
	}else{ //Simulação na CPU

		if (*CSOUR0_.JOBEND != 0)
			goto L103;

L101:;
		//Simulação de uma nova ducha e pontuação.
		shower2_();
		if (*CSOUR0_.JOBEND != 0)
			goto L102;

		timer2_(*CNTRL_.TSEC);

		//Terminar a simulação após o tempo previsto ou após completar Chuveiros DSHN.
		if ((*CNTRL_.TSEC < *CNTRL_.TSECA) && (*CNTRL_.SHN < *CNTRL_.DSHN)) {
			//Escreva os resultados parciais após cada período de despejo.
			if (*CDUMP_.LDUMP) {
				if (*CNTRL_.TSEC - *CNTRL_.TSECAD > *CNTRL_.DUMPP) {
					*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
					pmwrt2_(-1);
					printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
					*CNTRL_.TSECAD = *CNTRL_.TSEC;
					*CNTRL_.CPUT0 = cputim2_();
					goto L101;
				}
			}
			goto L101;
		}
L102:;
		*CNTRL_.TSIM = *CNTRL_.TSIM + cputim2_() - *CNTRL_.CPUT0;
	}

L103:;//Imprimir resultados Finais
	pmwrt2_(1);
	printf("  Number of simulated showers = %.6E\n", *CNTRL_.SHN);
	plotdose2_();
	memoryFreeCPU();
	printf("  *** END ***\n");
	return 0;
	
}

